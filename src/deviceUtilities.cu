#include <iostream>

using namespace std;

void queryDeviceInformation(void) {
	hipDeviceProp_t prop;

	int count;
	hipGetDeviceCount(&count);

	for (int i=0; i< count; i++) {

		hipGetDeviceProperties(&prop, i);

		cout << "--- General Information for device " << i << endl << endl;

		cout << "Name: " << prop.name << endl;
		cout << "Compute capability: " << prop.major << "." << prop.minor << endl;
		cout << "Clock rate: " << prop.clockRate << endl;

		cout << "Device copy overlap: ";
			if (prop.deviceOverlap)
				cout << "Enabled " << endl;
			else
				cout << "Disabled" << endl;

		cout << "Kernel execution timeout: ";
			if (prop.kernelExecTimeoutEnabled)
				cout << "Enabled " << endl;
			else
				cout << "Disabled" << endl;

		cout << endl;
		cout << "--- Memory Information for device " << i << endl << endl;

		cout << "Total global memory: " << prop.totalGlobalMem << endl;
		cout << "Total constant memory: " << prop.totalConstMem << endl;
		cout << "Max mem pitch: " << prop.memPitch << endl;
		cout << "Texture Alignment: " << prop.textureAlignment << endl;
	}
}
