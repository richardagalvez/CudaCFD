#include <iostream>

using namespace std;

void queryDeviceInformation(void) {
	hipDeviceProp_t prop;

	int count;
	hipGetDeviceCount(&count);

	for (int i=0; i< count; i++) {

		hipGetDeviceProperties(&prop, i);

		cout << "--- General Information for device " << i << endl << endl;

		cout << "Name: " << prop.name << endl;
		cout << "Compute capability: " << prop.major << "." << prop.minor << endl;
		cout << "Clock rate: " << prop.clockRate << endl;

		cout << "Device copy overlap: ";
			if (prop.deviceOverlap)
				cout << "Enabled " << endl;
			else
				cout << "Disabled" << endl;

		cout << "Kernel execution timeout: ";
			if (prop.kernelExecTimeoutEnabled)
				cout << "Enabled " << endl;
			else
				cout << "Disabled" << endl;

		cout << endl;
		cout << "--- Memory Information for device " << i << endl << endl;

		cout << "Total global memory: " << prop.totalGlobalMem << endl;
		cout << "Total constant memory: " << prop.totalConstMem << endl;
		cout << "Max mem pitch: " << prop.memPitch << endl;
		cout << "Texture Alignment: " << prop.textureAlignment << endl;

		cout << endl;
		cout << "--- MP Information for device " << i << endl << endl;

		cout << "Multiprocessor count: " << prop.multiProcessorCount << endl;
		cout << "Shared memory per block: " << prop.sharedMemPerBlock << endl;
		cout << "Registers per block: " << prop.regsPerBlock << endl;
		cout << "Threads in warp: " << prop.warpSize << endl;
		cout << "Max threads per block: " << prop.maxThreadsPerBlock << endl;
		cout << "Max thread dimensions: " << prop.maxThreadsDim[0] << ", "
			 << prop.maxThreadsDim[1] << ", " << prop.maxThreadsDim[2] << endl;
		cout << "Max grid dimensions: " << prop.maxGridSize[0] << ", "
					 << prop.maxGridSize[1] << ", " << prop.maxGridSize[2] << endl;

	}
}
